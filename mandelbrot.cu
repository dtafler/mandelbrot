#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include <hip/hip_runtime.h>

const int WIDTH = 800;
const int HEIGHT = 600;

__device__ int mandelbrot(double real, double imag, int maxIter) {
    double zr = 0.0, zi = 0.0;
    int iter = 0;

    while (zr * zr + zi * zi <= 4.0 && iter < maxIter) {
        double temp = zr * zr - zi * zi + real;
        zi = 2.0 * zr * zi + imag;
        zr = temp;
        iter++;
    }

    return iter;
}

__global__ void mandelbrotKernel(int width, int height, double centerX, double centerY, double scale, int maxIter, int* d_image) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        double real = scale * (x - (width / 2.0)) - centerX;
        double imag = centerY + scale * (y - (height / 2.0));

        int iter = mandelbrot(real, imag, maxIter);
        d_image[y * width + x] = iter;
    }
}

sf::Color getColor(int iter, int maxIter) {
    if (iter == maxIter) return sf::Color::Black;
    double t = (double)iter / maxIter;
    return sf::Color((int)(9 * (1 - t) * t * t * t * 255),
                     (int)(15 * (1 - t) * (1 - t) * t * t * 255),
                     (int)(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255));
}
int getMaxIterations(double zoom) {
    return (int)(100 + log2(zoom) * 50);  // Increase iterations as zoom deepens
}

int main() {
    sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Mandelbrot Explorer with CUDA");

    int* d_image;
    hipMalloc(&d_image, WIDTH * HEIGHT * sizeof(int));
    int* h_image = new int[WIDTH * HEIGHT];

    double centerX = -0.75, centerY = 0.0, scale = 4.0 / WIDTH;
    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);

    sf::Image image;
    image.create(WIDTH, HEIGHT);
    sf::Texture texture;
    sf::Sprite sprite(texture);

    sf::Font font;
    if (!font.loadFromFile("/usr/share/fonts/truetype/dejavu/DejaVuSans.ttf")) {
        std::cerr << "Error loading font!\n";
    }

    sf::Text zoomText;
    zoomText.setFont(font);
    zoomText.setCharacterSize(20);
    zoomText.setFillColor(sf::Color::White);

    while (window.isOpen()) {
        zoomText.setString("Zoom: " + std::to_string(1.0 / scale) + "\nCenter: (" +
                        std::to_string(centerX) + ", " + std::to_string(centerY) + ")");
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed)
                window.close();
            
            bool redraw = false;

            // handle zoom
            double zoomSpeed = 0.25;
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::D)) {
                scale *= (1.0 - zoomSpeed);
                redraw = true;
            }
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::F)) {
                scale *= (1.0 + zoomSpeed);
                redraw = true;
            }

            // handle pan
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::Up)) {
                // std::cout << "UP\n";
                centerY -= 50 * scale;
                redraw = true;
            }
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::Down)) {
                // std::cout << "DOWN\n";
                centerY += 50 * scale;
                redraw = true;
            }
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::Left)) {
                // std::cout << "LEFT\n";
                centerX += 50 * scale;
                redraw = true;
            }
            if (sf::Keyboard::isKeyPressed(sf::Keyboard::Right)) {
                // std::cout << "RIGHT\n";
                centerX -= 50 * scale;
                redraw = true;
            }

            if (redraw) {
                int maxIter = getMaxIterations(1.0 / scale);
                // std::cout << maxIter << std::endl;
                mandelbrotKernel<<<gridSize, blockSize>>>(WIDTH, HEIGHT, centerX, centerY, scale, maxIter, d_image);
                hipMemcpy(h_image, d_image, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);
                
                for (int x = 0; x < WIDTH; ++x) {
                    for (int y = 0; y < HEIGHT; ++y) {
                        image.setPixel(x, y, getColor(h_image[y * WIDTH + x], maxIter));
                    }
                }
            }

        }

        texture.loadFromImage(image);
        sprite.setTexture(texture, true);

        window.clear();
        window.draw(sprite);
        window.draw(zoomText);
        window.display();
    }

    hipFree(d_image);
    delete[] h_image;
    return 0;
}
